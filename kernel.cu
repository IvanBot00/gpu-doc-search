#include "hip/hip_runtime.h"
#include <stdio.h>
#include "defines.h"


__global__ void docFrequencyKernel(unsigned *output, unsigned *input, unsigned numDocs) {

    __shared__ unsigned private_df;

    int t = threadIdx.x;

    if (t == 0)
	private_df = 0;

    //__syncthreads;

    int i = t + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < numDocs) {
	if (input[i] > 0) // maybe check if within bounds?
	    atomicAdd(&private_df, 1);
	i += stride;
    }

   // __syncthreads;

    if (threadIdx.x == 0) {
	atomicAdd(output, private_df);
    }
}


void calculateDocFrequency(unsigned *df_d, unsigned *tf_d, const unsigned *tf_h, const unsigned numWords, const unsigned numDocs) {
    
    int BLOCK_SIZE = 256;
    
    dim3 dimGrid((numDocs - 1) / BLOCK_SIZE + 1, 1 , 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    hipStream_t stream0;
    hipStream_t stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    size_t rowSize = numDocs * sizeof(unsigned);

    unsigned *doc0, *doc1;

    for (unsigned i = 0; i < numWords; i += 2) {
	doc0 = tf_d + (i * numDocs);
	doc1 = tf_d + ((i+1) * numDocs);

	hipMemcpyAsync(doc0, tf_h + i * numDocs, rowSize, hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(doc1, tf_h + (i + 1) * numDocs, rowSize, hipMemcpyHostToDevice, stream1);

	docFrequencyKernel<<<2, BLOCK_SIZE, 0, stream0>>>(&df_d[i], doc0, numDocs);
	docFrequencyKernel<<<2, BLOCK_SIZE, 0, stream1>>>(&df_d[i+1], doc1, numDocs);
    }
}

__global__ void bm25Kernel(float *output, const unsigned *tf, const unsigned numWords, const unsigned numDocs) {

    unsigned Col = blockDim.x * blockIdx.x + threadIdx.x;

    float docScore = 0.0;

    for (int Row = 0; Row < numWords; ++Row) {
	int df = 10; //docFrequency(tf, Row, numDocs);
	int doctf = tf[Row * numDocs + Col];
	docScore += logf((numDocs - df + 0.5) / (df + 0.5)) * ((K_1 + 1) * doctf / (K + doctf));	
    } 

    __syncthreads;

    if (Col < numDocs) {
	output[Col] = docScore;
    }
}

void calculateBM25(float *output, const unsigned *tf, const unsigned numWords, const unsigned numDocs) {

    int BLOCK_SIZE = 512;

    dim3 dimGrid((numDocs - 1) / BLOCK_SIZE + 1, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    bm25Kernel<<<dimGrid, dimBlock>>>(output, tf, numWords, numDocs);
}
/*
__device__ float dotProduct(float *a, float *b, int size) {

}

__device__ void cosineSimilarity() {

    // calculate magntidue

    // calc dot product

    // calc dot proudct / magntiude

    // set result to array index 

}

void scoreDocuments(const int *tf, const float *idf, const int numWords, const int numDocs) {

    dim3 dimGrid = 4;
    dim3 dimBlock = 512;

    for (unsigned int i = 0; i < num_docs; i += 2) {

	// stream = new cudastream
	// stream 2 = new cudastream

	// memcpy async

	
     }
}
*/
/*
__global__ void calculateIDF(int size, int d, int w, const float *m, const float *idf, const float *buffer) {

    int i = threadIdx.x + blockId.x * blockDim.x;

    int stride = blockDim.x * gridDim.x;

    while (i < size) {
	int tf = m[i];
	if (tf > 0) {
	    // atomic add to the buffer
	}
	i += stride;
    }

    if (threadIdx.x < w) {
	//atomic add idf buffer to idf array
    }
}

void launchIDF(int size, int d, int w, const float *m, const float *idf) {
    
    dim3 dimGrid();
    dim3 dimBlock();

    calculateIDF<<<dimGrid, dimBlock>>>(d, w, m, idf);
}
*/

/*
__global__ void calculateScore() {
    
    // dot product
    for (int i = 0; i < numWords; ++i) {
	result[i] += tfidf[i] * q[i]
    }   
}

void launchScore(int size, int numDocs, int numWords, const float *m, const float *q) {

    SegSize = 1024 * numWords;

    dim3 DimGrid = SegSize / 256;
    dim3 DimBlock = 256;

    // load query into global mem

    // Use cuda streams to calculate document score
    for (int i = 0; i < size; i += SegSize * 2) {
	hipMemcpyAsync(m0_d, m + i, SegSize * sizeof(float), hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(m1_d, m + i + SegSize, SegSize * sizeof(float), cudaMemCpyHostToDevice, stream1);

	calculateScore<<<DimGrid, DimBlock, 0, stream0>>>(m0_d, SegSize, numDocs, numWords);
	calculateScore<<<DimGrid, DimBlock, 0, stream1>>>(m1_d, SegSize, numDocs, numWords);

	hipMemcpyAsync(results_h + i, results0_d, SegSize * sizeof(float), cudaMemCpyDeviceToHost, stream0);
	hipMemcpyAsync(results_h + i + SegSize, results1_d, SegSize * sizeof(float), cudaMemCpyDeviceToHost, stream1);
    }
}
*/
