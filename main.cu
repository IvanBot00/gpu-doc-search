#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "support.h"
#include "kernel.cu"

int main (int argc, char *argv[])
{
    hipError_t cuda_ret;

    srand(time(NULL));

    // Allocate host variables
    unsigned *tf_h, *tf_d, *df_h, *df_d;
    float *scores_h, *scores_d;
    size_t tf_sz, scores_sz, df_sz;
    unsigned numDocs, numWords;

    if (argc == 1) {
	numDocs = 1000;
	numWords = 1000;
    } else if (argc == 2) {
	numDocs = atoi(argv[1]);
	numWords = atoi(argv[1]);
	printf("%u", numDocs);
    } else if (argc == 3) {
	numWords = atoi(argv[1]);
	numDocs = atoi(argv[2]);
    } else if (argc == 4) {
	printf("File read not implemented yet\n");
    } else {
	printf("\n    Invalid input parameters!");
    }
    
    tf_sz = numDocs * numWords;
    scores_sz = numDocs;
    df_sz = numWords;

    tf_h = (unsigned*) malloc(sizeof(unsigned) * tf_sz);
    for (unsigned int i=0; i < tf_sz; ++i) {
	unsigned initialVal = rand() % 2;
	tf_h[i] = initialVal * rand() % 9;
    }

    df_h = (unsigned*) malloc(sizeof(unsigned) * df_sz);
    for (unsigned int i=0; i < df_sz; ++i) { df_h[i] = 0; }

    scores_h = (float*) malloc(sizeof(float) * scores_sz);
    for (unsigned int i=0; i < scores_sz; ++i) { scores_h[i] = 0; }

	
    // Allocate device variables
    printf("Allocating device variables..."); fflush(stdout);
    
    hipMalloc((void**) &tf_d, sizeof(unsigned) * tf_sz);
    hipMalloc((void**) &scores_d, sizeof(float) * scores_sz);
    hipMalloc((void**) &df_d, sizeof(unsigned) * df_sz);

    hipDeviceSynchronize();

    // Copy host variables to device
    printf("Copying data from host to device..."); fflush(stdout);

   // hipMemcpy(tf_d, tf_h, tf_sz * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemset(df_d, 0, numWords * sizeof(unsigned));
    hipMemcpy(scores_d, scores_h, scores_sz * sizeof(float), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    // Launch Kernels ----------

    calculateDocFrequency(df_d, tf_d, tf_h, numWords, numDocs);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    calculateBM25(scores_d, tf_d, df_d, numWords, numDocs);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    // -------------------------
    
    // Copy device variables to host

    hipMemcpy(scores_h, scores_d, scores_sz * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(df_h, df_d, df_sz * sizeof(unsigned), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Verify results

    printf("Verifying Results..."); fflush(stdout);

    verify_df(df_h, tf_h, numWords, numDocs);
    verify_bm25(scores_h, tf_h, df_h, numWords, numDocs);


    // Free memory
    free(tf_h);
    free(scores_h);
    free(df_h);

    hipFree(tf_d);
    hipFree(scores_d);
    hipFree(df_d);
}

